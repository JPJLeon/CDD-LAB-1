#include <iostream>
#include <hip/hip_runtime.h>
#define xd 0.001
#define STEPS 10

/*  Lectura Archivo */
void Read(float **f, int *M, int *N, int tipo=0) {
    FILE *fp;
    fp = fopen("initial.txt\0", "r");
    fscanf(fp, "%d %d\n", M, N);
		float *f1;

    int size = (*M) * (*N);
		if (tipo == 0)
    	f1 = new float[size];
		else
			hipHostMalloc(&f1, sizeof(float)* size, hipHostMallocDefault); //pinned memory
	
		for(int i = 0; i < size; i++){
					fscanf(fp, "%f ", &(f1[i]));
					// printf("%d ", i*4 + x);
		}
	
    fclose(fp);
    *f = f1;
}



/*  Escritura de archivo initial con array */
void Write(int *f, int M, int N, const char *filename) {
    FILE *fp;
    fp = fopen(filename, "w");
    fprintf(fp, "%d %d\n", N, M);
    int Largo = M*N;

		for(int i = 0; i < Largo-1; i++){
				fprintf(fp, "%d ", f[i]);
			
		}
		fprintf(fp, "%d\n", f[Largo-1]);
    //printf("\n");
    fclose(fp);
}


//funcion auxiliar %, funciona con entradas negativas
__host__ __device__ int modulo(int a, int b){
    //a%b
    if (a >= 0){
        return a %b;
    }
    return b + a;
}

void imprimir_malla(float *f, int N , int M){
		for(int j = 0; j< M; j ++){
			for(int i = 0; i< N; i ++){
						printf("%.2f ", f[i+ j*M]);
				}
				printf("\n");
		}
		printf("-----\n");
}

float* concatenar(float *f1, float *f2, float *f3, float *f4, int N, int M){
		float *resultado = new float[N*M];
		int size = N * M /4;
		for (int i = 0; i < size; i ++){
				resultado[i + 0*size] = f1[i];
				resultado[i + 1*size] = f1[i];
				resultado[i + 2*size] = f1[i];
				resultado[i + 3*size] = f1[i]; 
		}
		return resultado;
}

/* Procesamiento CPU */
void CPU_1_step(float *f_in, float *f_out, int N, int M){
		int x,y;
		for (int i = 0; i < N*M; i++){
				x = i % N;
				y = i / N;
				f_out[i] = (f_in[modulo(x+1, N) + y*N] + f_in[modulo(x-1, N) + y*N]) /(2); // xd
				//f_out[i] = 2;
		}
}

void CPU(){
	int M, N;
	float *f_in, *f_out, *temp;
	clock_t t1, t2;
	double ms;

	Read(&f_in, &M, &N,0);
	//imprimir_malla(f_in, N, M);
	f_out = new float[N*M];

	t1 = clock();
	for(int step = 0; step< STEPS; step++){
			CPU_1_step(f_in, f_out, N,M);
			temp = f_out;
			f_out = f_in;
			f_in = temp;
	}
	f_out =f_in;
	t2 = clock();	
	ms = 1000.0 * (double)(t2 - t1) / CLOCKS_PER_SEC;
	printf("Tiempo CPU: %f[ms]\n", ms);

	//imprimir_malla(f_out, N,M);
 	delete[] f_in;
 	delete[] f_out;
}



/*  Procesamiento GPU, 1 stream */
__global__ void kernel_1(float *f, float *f_out, int N, int M){
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid < M){ //1 thread para cada fila
	    float anterior = f[modulo(-1, N) + tid*N];
			float actual = f[0 + tid*N];
			float siguiente; 
			for (int i = 0; i< N; i++){
					siguiente = f[modulo(i+1, N) + tid*N];
					f_out[i + tid*N] = (anterior + siguiente) / (2); //xd

					anterior = actual;
					actual = siguiente;
			}
	}
}
void GPU_1_stream(){	
		printf("gpu 1\n");
		hipEvent_t ct1, ct2;	
		float dt;
		int M, N;
		float *f_host, *f_hostout, *f, *f_out, *temp;
		int gs, bs = 256;

		Read(&f_host, &M, &N,0);
		gs = (int)ceil((float) M / bs);    
		//imprimir_malla(f_host, N,M);

		hipMalloc((void**)&f, M * N * sizeof(float));
		hipMemcpy(f, f_host, M * N * sizeof(float), hipMemcpyHostToDevice);
		hipMalloc((void**)&f_out, M * N * sizeof(float));
		//cudaMalloc((void**)&temp, M * N * sizeof(float));

		hipEventCreate(&ct1);
		hipEventCreate(&ct2);
		hipEventRecord(ct1);


		// llamadas al kernel
		for (int i = 0 ; i< STEPS; i++){
			kernel_1<<<gs, bs>>>(f, f_out, N, M);
			temp = f_out;
			f_out = f;
			f = temp;
		}
		f_out =f;
	
	
		hipEventRecord(ct2);
		hipEventSynchronize(ct2);
		hipEventElapsedTime(&dt, ct1, ct2);
		f_hostout = new float[M * N];
		hipMemcpy(f_hostout, f, M * N * sizeof(float), hipMemcpyDeviceToHost);

		//Write(f_hostout, M, N, "initial_S.txt\0");
		//imprimir_malla(f_hostout, N,M);
		std::cout << "Tiempo " << ": " << dt << "[ms]" << std::endl;
		hipFree(f);
		//cudaFree(temp);
		hipFree(f_out);
		delete[] f_host;
		delete[] f_hostout;
}



__global__ void kernel_2(float *f, float *f_out, int N, int M){
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid < M/4){ //1 thread para cada fila
	    float anterior = f[modulo(-1, N) + tid*N];
			float actual = f[0 + tid*N];
			float siguiente; 
			for (int i = 0; i< N; i++){
					siguiente = f[modulo(i+1, N) + tid*N];
					f_out[i + tid*N] = (anterior + siguiente) / 2; //xd
					anterior = actual;
					actual = siguiente;
			}
	}
}


/*  Procesamiento GPU, 4 stream horizontal*/
void GPU_4_stream_horizontal(){
		printf("stream horizontal \n");
		hipEvent_t ct1, ct2;	
		float dt;
		int M, N;
		float *f_host;
		float * f_in;
		float *f_out;
		int gs, bs = 256;

		//crear streams
		hipStream_t str1, str2, str3, str4;
		hipStreamCreate(&str1);
		hipStreamCreate(&str2);
		hipStreamCreate(&str3);
		hipStreamCreate(&str4);

		Read(&f_host, &M, &N,1);
		gs = (int)ceil((float) (M/4) / bs);    
		//imprimir_malla(f_host, N,M);
		int size =  M/4 * N ;

		hipMalloc(&f_in,  M * N* sizeof(float));
		hipMalloc(&f_out, M * N* sizeof(float));
		float *out = new float[N*M];
		float *temp;

		//host to device
		hipMemcpyAsync(&f_in[size*0], &f_host[size*0], size * sizeof(float), hipMemcpyHostToDevice, str1);
		hipMemcpyAsync(&f_in[size*1], &f_host[size*1], size * sizeof(float), hipMemcpyHostToDevice, str2);
		hipMemcpyAsync(&f_in[size*2], &f_host[size*2], size * sizeof(float), hipMemcpyHostToDevice, str3);
		hipMemcpyAsync(&f_in[size*3], &f_host[size*3], size * sizeof(float), hipMemcpyHostToDevice, str4);
		

		//kernel calls
		hipEventCreate(&ct1);
		hipEventCreate(&ct2);
		hipEventRecord(ct1);
	
		// llamadas al kernel
		for (int i = 0 ; i< STEPS; i++){
			kernel_2<<<gs, bs,0,str1>>>(f_in, f_out, N, M);
			kernel_2<<<gs, bs,0,str2>>>(&f_in[size*1], &f_out[size*1], N, M);
			kernel_2<<<gs, bs,0,str3>>>(&f_in[size*2], &f_out[size*2], N, M);
			kernel_2<<<gs, bs,0,str4>>>(&f_in[size*3], &f_out[size*3], N, M);
			temp = f_out;
			f_out = f_in;
			f_in = temp;
		}
		//f_out =f_in;
		
	
	
		hipEventRecord(ct2);
		hipEventSynchronize(ct2);
		hipEventElapsedTime(&dt, ct1, ct2);


		//device to host
		hipMemcpyAsync(&out[size*0], &f_in[size*0], size * sizeof(float), hipMemcpyDeviceToHost,str1);
		hipMemcpyAsync(&out[size*1], &f_in[size*1], size * sizeof(float), hipMemcpyDeviceToHost,str2);
		hipMemcpyAsync(&out[size*2], &f_in[size*2], size * sizeof(float), hipMemcpyDeviceToHost,str3);
		hipMemcpyAsync(&out[size*3], &f_in[size*3], size * sizeof(float), hipMemcpyDeviceToHost,str4);

		//Write(out, M, N, "initial_S.txt\0");
		hipDeviceSynchronize();
		//imprimir_malla(out, N,M);
		std::cout << "Tiempo " << ": " << dt << "[ms]" << std::endl;
		hipFree(f_host);
		hipFree(f_in);
		hipFree(f_out);
}



/*  Procesamiento GPU, 4 stream vertical*/
void GPU_4_stream_vertical(){
			printf("gpu 1");
}
__global__ void kernel3(int *f, int *f_out, int X, int N, int M){
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid < M*N){
	}
}


//--------------------------------------------------------------------------------


/*  Codigo Principal */
int main(int argc, char **argv){
		
	//ejecucion cpu
	//CPU(); //212

	GPU_1_stream(); //23 1784

	//GPU_4_stream_horizontal(); //23 1442

	//GPU_4_stream_vertical();
	
	return 0;
}