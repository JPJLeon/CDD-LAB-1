// Basado en el codigo entregado durante clases practicas

#include <iostream>
#include <time.h>
#include <hip/hip_runtime.h>

/*
 *  Lectura Archivo
 */
void Read(float** R, float** G, float** B, int *N, int *S, int** positions, const char *filename) {
    printf("Leemos el archivo %s!\n", filename); 
    FILE *fp;
    fp = fopen(filename, "r");
    fscanf(fp, "%d %d\n", N, S);
    // obtenemos segunda linea con nuevas posiciones
    int P = (*N) / (*S);
    int *positions1 = new int[P*P];
    for(int i = 0; i < P*P; i++){
	    fscanf(fp, "%*d ", &positions[i]);
    }
	
    
    int imsize = (*N) * (*N);
    float* R1 = new float[imsize];
    float* G1 = new float[imsize];
    float* B1 = new float[imsize];
	for(int i = 0; i < imsize; i++)
	    fscanf(fp, "%f ", &(R1[i]));
	for(int i = 0; i < imsize; i++)
	    fscanf(fp, "%f ", &(G1[i]));
	for(int i = 0; i < imsize; i++)
	    fscanf(fp, "%f ", &(B1[i]));
    fclose(fp);

    *R = R1; *G = G1; *B = B1; *positions = positions1;
}

/*
 *  Escritura Archivo
 */
void Write(float* R, float* G, float* B, 
	       int M, int N, const char *filename) {
    FILE *fp;
    fp = fopen(filename, "w");
    fprintf(fp, "%d %d\n", M, N);
    for(int i = 0; i < M*N-1; i++)
        fprintf(fp, "%f ", R[i]);
    fprintf(fp, "%f\n", R[M*N-1]);
    for(int i = 0; i < M*N-1; i++)
        fprintf(fp, "%f ", G[i]);
    fprintf(fp, "%f\n", G[M*N-1]);
    for(int i = 0; i < M*N-1; i++)
        fprintf(fp, "%f ", B[i]);
    fprintf(fp, "%f\n", B[M*N-1]);
    fclose(fp);
}


__host__ __device__ int findNewPosition(int P, int position, int* positions){

	int newPosition;

	for (int j = 0; j < P*P; j++) {
    	if(position == positions[j]) {
        	newPosition = j;
        	break;
    	}
	}

	return newPosition;
}

/*
 *  Procesamiento Imagen CPU
 */
void funcionCPU( float* R, float* G, float* B, float* Rout, float* Gout, float* Bout, int N, int S, int* positions){
	int P = N/S;
	//cambiar
	for (int i = 0; i < N*N; i++){
		int x = i % N;
		int y = i / N;
		int newX = x / P;
		int newY = y / P;
		int position = newX + newY * P;
		int newPosition = findNewPosition(P, position, positions);
		int newI;

		newI = (i + S*S*(newPosition - position));

		Rout[newI] = R[i];
		Gout[newI] = G[i];
		Bout[newI] = B[i];
	}
}

/*
 *  Procesamiento Imagen GPU
 */
__global__ void kernelGPU( float* R, float* G, float* B, float* Rout, float* Gout, float* Bout, int N, int S, int* positions){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int P = N/S;

	if(i < N*N){
		int x = i % N;
		int y = i / N;
		int newX = x / P;
		int newY = y / P;
		int position = newX + newY * P;
		int newPosition = findNewPosition(P, position, positions);
		int newI;

		newI = (i + S*S*(newPosition - position));

		Rout[newI] = R[i];
		Gout[newI] = G[i];
		Bout[newI] = B[i];
	} 
}

/*
 *  Codigo Principal
 */
int main(int argc, char **argv){

    /*
     *  Inicializacion
     */
	clock_t t1, t2;
	hipEvent_t ct1, ct2;
	double ms;
	float dt;
	int N, S;
	int *positions;
    float *Rhost, *Ghost, *Bhost;
    float *Rhostout, *Ghostout, *Bhostout;
    float *Rdev, *Gdev, *Bdev;
    float *Rdevout, *Gdevout, *Bdevout;
    
    //cambiar con los nombres de los archivos correctos
    char names[5][3][30] = {{"img100x100.txt\0", "img100x100CPU.txt\0", "img100x100GPU.txt\0"}, 
    	{"img200x200.txt\0", "img200x200CPU.txt\0", "img200x200GPU.txt\0"},
    	{"img400x400.txt\0", "img400x400CPU.txt\0", "img400x400GPU.txt\0"},
    	{"img800x800.txt\0", "img800x800CPU.txt\0", "img800x800GPU.txt\0"},
    	{"img1600x1600.txt\0", "img1600x1600CPU.txt\0", "img1600x1600GPU.txt\0"}};

    for (int i=0; i<5; i++){
	    Read(&Rhost, &Ghost, &Bhost, &N, &S, &positions, names[i][0]); //leemos archivo y reservamos memoria

	    /*
	     *  Parte CPU
	     */
	    Rhostout = new float[N*N];	//reservamos memoria
	    Ghostout = new float[N*N];
	    Bhostout = new float[N*N];

	    t1 = clock();
	    funcionCPU(Rhost, Ghost, Bhost, Rhostout, Ghostout, Bhostout, N, S, positions); // Agregar parametros!
	    t2 = clock();
	    ms = 1000.0 * (double)(t2 - t1) / CLOCKS_PER_SEC;
	    std::cout << "Tiempo CPU: " << ms << "[ms]" << std::endl;
	    Write(Rhostout, Ghostout, Bhostout, N, N, names[i][1]);

	    delete[] Rhostout; delete[] Ghostout; delete[] Bhostout;
	    
	    /*
	     *  Parte GPU
	     */

	    int grid_size, block_size = 256;
	    grid_size = (int)ceil((float) N * N / block_size);
	        
	    hipMalloc((void**)&Rdev, N * N * sizeof(float));
	    hipMalloc((void**)&Gdev, N * N * sizeof(float));
	    hipMalloc((void**)&Bdev, N * N * sizeof(float));
	    hipMemcpy(Rdev, Rhost, N * N * sizeof(float), hipMemcpyHostToDevice);
	    hipMemcpy(Gdev, Ghost, N * N * sizeof(float), hipMemcpyHostToDevice);
	    hipMemcpy(Bdev, Bhost, N * N * sizeof(float), hipMemcpyHostToDevice);
	        
	    hipMalloc((void**)&Rdevout, N * N * sizeof(float));
	    hipMalloc((void**)&Gdevout, N * N * sizeof(float));
	    hipMalloc((void**)&Bdevout, N * N * sizeof(float));
	    
	    hipEventCreate(&ct1);
	    hipEventCreate(&ct2);
	    hipEventRecord(ct1);
	    kernelGPU<<<grid_size, block_size>>>(Rdev, Gdev, Bdev, Rdevout, Gdevout, Bdevout, N, S, positions); // Agregar parametros!
	    hipEventRecord(ct2);
	    hipEventSynchronize(ct2);
	    hipEventElapsedTime(&dt, ct1, ct2);
	    std::cout << "Tiempo GPU: " << dt << "[ms]" << std::endl;

	    Rhostout = new float[N*N];
	    Ghostout = new float[N*N];
	    Bhostout = new float[N*N];
	    hipMemcpy(Rhostout, Rdevout, N * N * sizeof(float), hipMemcpyDeviceToHost);
	    hipMemcpy(Ghostout, Gdevout, N * N * sizeof(float), hipMemcpyDeviceToHost);
	    hipMemcpy(Bhostout, Bdevout, N * N * sizeof(float), hipMemcpyDeviceToHost);
	    Write(Rhostout, Ghostout, Bhostout, N, N, names[i][2]);

	    hipFree(Rdev); hipFree(Gdev); hipFree(Bdev);
    	hipFree(Rdevout); hipFree(Gdevout); hipFree(Bdevout);
    	delete[] Rhost; delete[] Ghost; delete[] Bhost;
    	delete[] Rhostout; delete[] Ghostout; delete[] Bhostout;
	}
	return 0;
}