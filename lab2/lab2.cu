#include <iostream>
#include <hip/hip_runtime.h>

/*  Lectura Archivo */
void Read(int **f, int *M, int *N, const char *filename, int X, int tipo) {
    FILE *fp;
    fp = fopen(filename, "r");
    fscanf(fp, "%d %d\n", N, M);

    int imsize = (*M) * (*N) * X;
    int* f1 = new int[imsize];
    int Largo = (*M) * (*N);

    if (tipo != 0){ // AoS
		for(int x=0; x<X; x++){
			for(int i = 0; i < Largo; i++){
	        	fscanf(fp, "%d ", &(f1[i*4 + x]));
		        // printf("%d ", i*4 + x);
			}
	    }

	    // Datos M = 6, N = 4

	    //  0  1  2  3  4  5  6  7  8  9 10 11 12 13 14 15 16 17 18 19 20 21 22 											23
	    // 24 25 26 27 28 29 30 31 32 33 34 35 36 37 38 39 40 41 42 43 44 45 46 											47
	    // 48 49 50 51 52 53 54 55 56 57 58 59 60 61 62 63 64 65 66 67 68 69 70 											71
	    // 72 73 74 75 76 77 78 79 80 81 82 83 84 85 86 87 88 89 90 91 92 93 94 											95
	    
	    //  0 24 48 72  1 25 49 73  2 26 50 74  3 27 51 75  4 28 52 76   5 29 53 77  6 30 54 78  7 31 55 79  8 32 56 80 ... 95

	    //  0  0  0  0  0  0  0  0  0  0  0  0  0  0  0  0  0  0  0  0  0  0  0  0  1  1  1  1  1  1  1  1  1 ..    		 3	(x) < X
	    //  0  1  2  3  4  5  6  7  8  9 10 11 12 13 14 15 16 17 18 19 20 21 22 23  0  1  2  3  4  5  6  7  8 ...			23 (i) < N*M

	    //  i*4 + x
	    //  0  4  8 12 16 20 24 28 		....  		1  5  9 13 17 21 25 29    ......	2  6 10 14 18 22 	....	3  7 11 15 19 .. 

	} else{ // SoA 
		for(int j=0; j<X; j++){
	    	for(int i = 0; i < Largo; i++){
		        fscanf(fp, "%d ", &(f1[i + j*Largo]));
		        //printf("%d ", f1[i + j*Largo]);
	    	}
		    // fscanf(fp, "%d\n", &(f1[Largo-1 + j*Largo]));
	    	//printf("\n");
	    }
	}
    fclose(fp);
    *f = f1;
}

/*  Escritura de archivo initial con array */
void Write_AoS(int *f, int M, int N, const char *filename) {
    FILE *fp;
    fp = fopen(filename, "w");
    fprintf(fp, "%d %d\n", N, M);
    int Largo = M*N;
    for(int j=0; j<4; j++){
    	for(int i = 0; i < Largo-1; i++){
	        fprintf(fp, "%d ", f[i*4 + j]);
	    	//printf("%d ", f[i*4 + j]);
	    }
	    fprintf(fp, "%d\n", f[(Largo-1)*4 + j]);
	    //printf("%d\n", f[(Largo-1)*4 + j]);
    }
    printf("\n");
    fclose(fp);
}

/*  Escritura de archivo initial con array */
void Write_SoA(int *f, int M, int N, const char *filename) {
    FILE *fp;
    fp = fopen(filename, "w");
    fprintf(fp, "%d %d\n", N, M);
    int Largo = M*N;
    for(int j=0; j<4; j++){
    	for(int i = 0; i < Largo-1; i++){
	        fprintf(fp, "%d ", f[i + j*Largo]);
	    	//printf("%d ", f[i + j*Largo]);
	    }
	    fprintf(fp, "%d\n", f[Largo-1 + j*Largo]);
	    //printf("%d\n", f[Largo-1 + j*Largo]);
    }
    printf("\n");
    fclose(fp);
}

void validar(int *f, int N, int M){
	int suma=0;
	for(int i=0; i<N*M*4; i++){
		suma += f[i];
	}
	printf("Particulas: %d\n", suma);
}

//funcion auxiliar %, funciona con entradas negativas
__device__ int modulo(int a, int b){
    //a%b
    if (a >= 0){
        return a %b;
    }
    return b + a;
}

/*  Procesamiento GPU AoS Coalisiones */
__global__ void kernelAoS_col(int *f, int *f_out, int X, int N, int M){
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid < M*N){
		int idb = tid*4;
		int f0, f1, f2, f3;
		// Almacenamos los datos en memoria
		f0 = f[idb];
		f1 = f[idb+1];
		f2 = f[idb+2];
		f3 = f[idb+3];
		if(f0 && f2 && f1 == 0 && f3 == 0){
			f[idb] = 0;
			f[idb+1] = 1;
			f[idb+2] = 0;
			f[idb+3] = 1;
		} else if(f0 == 0 && f2 == 0 && f1 && f3){
			f[idb] = 1;
			f[idb+1] = 0;
			f[idb+2] = 1;
			f[idb+3] = 0;
		}
	}
}


/*  Procesamiento GPU AoS Streaming */
__global__ void kernelAoS_stream(int *f, int *f_out, int N, int M){
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid < N*M){
		// f0: der
		// f1: arr
		// f2: izq
		// f3: abj
		int x, y, idb;
		idb = tid*4;        //indice del f0 en el arreglo
		x = tid % M; // 4
		y = tid / M; // 1
		// Id de los nodos adyacentes
		int nd[] = {modulo(x+1,M)  + y              *M, 
					x              + modulo(y+1, N) *M, 
					modulo(x-1, M) + y              *M, 
					x              + modulo(y-1, N) *M };
		// Recorremos las direcciones
		for(int i=0; i<4; i++){
			// Seteo todas en 0
			//f_out[idb+i] = 0;
			// Si la particula se mueve en esta direccion
			if(f[idb+i] == 1){
				// La direccion del nodo de esa direccion cambia
				f_out[nd[i]*4+i] = 1;
			}
		}
	}
}


/*  Procesamiento GPU SoA Coalisiones */
__global__ void kernelSoA_col(int *f, int *f_out, int X, int N, int M){
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid < M*N){
		int f0, f1, f2, f3, Largo;
		Largo = N*M;
		// Almacenamos los datos en memoria
		f0 = f[tid];
		f1 = f[tid+1*Largo];
		f2 = f[tid+2*Largo];
		f3 = f[tid+3*Largo];
		if(f0 && f2 && f1 == 0 && f3 == 0){
			f[tid] = 0;
			f[tid+1*Largo] = 1;
			f[tid+2*Largo] = 0;
			f[tid+3*Largo] = 1;
		} else if(f0 == 0 && f2 == 0 && f1 && f3){
			f[tid] = 1;
			f[tid+1*Largo] = 0;
			f[tid+2*Largo] = 1;
			f[tid+3*Largo] = 0;
		}
	}
}

/*  Procesamiento GPU SoA Streaming */
__global__ void kernelSoA_stream(int *f, int *f_out, int X, int N, int M){
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid < N*M){
		// f0: der
		// f1: arr
		// f2: izq
		// f3: abj
		int x, y, Largo = N*M;
		x = tid % M; // 4
		y = tid / M; // 1
		// Id de los nodos adyacentes
		int nd[] = { modulo(x+1,M) + y*M,
					x + modulo(y+1,N)*M,
					modulo(x-1,M) + y*M,
					x + modulo(y-1,N)*M };
		// Recorremos las direcciones
		for(int i=0; i<X; i++){
			// Seteo todas en 0
			//f_out[tid + i*Largo] = 0;
			// Si la particula se mueve en esta direccion
			if(f[tid + i*Largo] == 1){
				// La direccion del nodo de esa direccion cambia
				f_out[nd[i] + i*Largo] = 1;
			}
		}
	}
}

__global__ void f_out_0(int *f_out, int N, int M){
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid < N*M*4){
    f_out[tid] = 0;
	}
}

//--------------------------------------------------------------------------------

//Pregunta 2, condiciones de borde con AoS

/*  Procesamiento GPU AoS Coalisiones */
__global__ void kernelAoS_col_borde(int *f, int *f_out, int X, int N, int M, int j){
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid < M*N){
		int idb = tid*4;
		int f0, f1, f2, f3, x ,y;
    x = tid % M; 
		y = tid / M; 

    // Almacenamos los datos en memoria
    f0 = f[idb+0];
    f1 = f[idb+1];
    f2 = f[idb+2];
    f3 = f[idb+3];

    bool borde =  (x == 0 || x == M -1 || y == 0 || y == N-1) ;
    bool horizontal = f0 && f2 && f1 == 0 && f3 == 0;
    bool vertical = f0 == 0 && f2 == 0 && f1 && f3;

    //if statement
    if (j == 0){
      if ( !borde ){ //si es que no se está en algun borde
        if(horizontal){
          f[idb] = 0;
          f[idb+1] = 1;
          f[idb+2] = 0;
          f[idb+3] = 1;
        } else if(vertical){
          f[idb] = 1;
          f[idb+1] = 0;
          f[idb+2] = 1;
          f[idb+3] = 0;
        }
      }
    }

    //operador ternario
    else if (j == 1){
        
        f[idb] = (borde ? f[idb] : 
                  (horizontal ? 0:
                   (vertical ? 1 : f[idb])));
        f[idb+1] = (borde ? f[idb +1 ] : 
                  (horizontal ? 1:
                   (vertical ? 0 : f[idb+ 1])));
        f[idb+2] = (borde ? f[idb +2] : 
                  (horizontal ? 0:
                   (vertical ? 1 : f[idb +2])));
        f[idb+3] = (borde ? f[idb +3] : 
                  (horizontal ? 1:
                   (vertical ? 0 : f[idb + 3])));
    }

    //operador booleano
    else if (j == 2){
        f[idb] =  (borde) * f[idb]  +  abs(borde -1)  * ((horizontal) * 0 + abs(horizontal-1) * ((vertical) * 1 + abs(vertical -1) * f[idb]));
        f[idb+ 1] =  (borde) * f[idb+ 1]  +  abs(borde -1)  * ((horizontal) * 1 + abs(horizontal-1) * ((vertical) * 0 + abs(vertical -1) * f[idb+ 1]));
        f[idb+ 2] =  (borde) * f[idb+ 2]  +  abs(borde -1)  * ((horizontal) * 0 + abs(horizontal-1) * ((vertical) * 1 + abs(vertical -1) * f[idb+ 2]));
        f[idb+ 3] =  (borde) * f[idb+ 3]  +  abs(borde -1)  * ((horizontal) * 1 + abs(horizontal-1) * ((vertical) * 0 + abs(vertical -1) * f[idb+ 3]));
    }
	}
}


/*  Procesamiento GPU AoS Streaming */
__global__ void kernelAoS_stream_borde(int *f, int *f_out, int N, int M, int j){
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid < N*M){
		int x, y, idb;
		idb = tid*4;        //indice del f0 en el arreglo
		x = tid % M; 
		y = tid / M; 
		// Id de los nodos adyacentes
		int nd[] = {modulo(x+1,M)  + y              *M, 
					      x              + modulo(y+1, N) *M, 
					      modulo(x-1, M) + y              *M, 
					      x              + modulo(y-1, N) *M };



		// if statement
    if (j == 0){
      for(int i=0; i<4; i++){
        // Seteo todas en 0
        f_out[idb+i] = 0;

        //condiciones de borde
        bool sur = (y == 0 && i==3);
        bool norte = (y == N-1 && i == 1);
        bool oeste = (x == 0 && i == 2);
        bool este = (x == M-1 && i == 0);
        // Si la particula se mueve en esta direccion
        if(f[idb+i] == 1){                               //si fi == 0
            if (sur){                         //si se mueve hacia abajo en el borde inferior
                f_out[nd[1] * 4 + 1] = 1;                //rebota hacia arriba 
            }
            else if (norte){
                f_out[nd[3] * 4 + 3] = 1;
            }
            else if (oeste){
                f_out[nd[0] *4 + 0] = 1;
            } 
            else if (este){
                f_out[nd[2] * 4 + 2] = 1;
            }
            else{
                f_out[nd[i]*4+i] = 1;      
            }
        }
      }
    }

    //operador ternario
    else if(j == 1){
      for(int i=0; i<4; i++){
        // Seteo todas en 0
        f_out[idb+i] = 0;

        bool sur = (y == 0 && i==3);
        bool norte = (y == N-1 && i == 1);
        bool oeste = (x == 0 && i == 2);
        bool este = (x == M-1 && i == 0);
        
        !(f[idb+i] == 1) ? true : 
            (sur) ?  f_out[nd[1] * 4 + 1] = 1 :
              (norte) ? f_out[nd[3] * 4 + 3] = 1:
                (oeste) ? f_out[nd[0] *4 + 0] = 1 : 
                  (este) ? f_out[nd[2] * 4 + 2] = 1 : f_out[nd[i]*4+i] = 1;
       
        }
      }

      //operador booleano
      else if (j == 2){
        for(int i=0; i<4; i++){
          // Seteo todas en 0
          f_out[idb+i] = 0;

          bool activo = (f[idb+i] == 1);
          bool sur = (y == 0 && i==3);
          bool norte = (y == N-1 && i == 1);
          bool oeste = (x == 0 && i == 2);
          bool este = (x == M-1 && i == 0);

          f_out[nd[1] * 4 + 1]  = activo * sur; 
          f_out[nd[3] * 4 + 3] = activo * abs(sur-1) * norte;
          f_out[nd[0] *4 + 0]   = activo * abs(sur-1) * abs(norte-1) * oeste; 
          f_out[nd[2] * 4 + 2]  = activo * abs(sur-1) * abs(norte-1) * abs(oeste -1) * este;
          f_out[nd[i]*4+i]      = activo * abs(sur-1) * abs(norte-1) * abs(oeste -1) * abs(este-1);
      }
    }
	}
}

/*  Codigo Principal */
int main(int argc, char **argv){
    /*
     *  Inicializacion
     */
	hipEvent_t ct1, ct2;
	float dt;
	// N eje y, M eje x
	int M, N;
    int *f_host, *f_hostout, *f, *f_out, *temp;
    char filename[15] = "initial.txt\0";
	int gs, bs = 256;
	int X = 4;

	// 2 metodos SoA y AoS
    for (int i=0; i<2; i++){
    	Read(&f_host, &M, &N, filename, X, i);

	    gs = (int)ceil((float) M * N * X / bs);    
	    hipMalloc((void**)&f, M * N * X * sizeof(int));
	    hipMemcpy(f, f_host, M * N * X * sizeof(int), hipMemcpyHostToDevice);
	    hipMalloc((void**)&f_out, M * N * X * sizeof(int));
	    hipMalloc((void**)&temp, M * N * X * sizeof(int));
    	validar(f_host, N, M);

	    hipEventCreate(&ct1);
	    hipEventCreate(&ct2);
	    hipEventRecord(ct1);

	    // Iteraciones de time step 
	    for (int j=0; j<1000; j++){
        f_out_0<<<gs, bs>>>(f_out, N, M);
	    	if (i == 0){
	    		kernelSoA_col<<<gs, bs>>>(f, f_out, X, N, M);
	    		kernelSoA_stream<<<gs, bs>>>(f, f_out, X, N, M);
	    	}
	    	else{
	    		kernelAoS_col<<<gs, bs>>>(f, f_out, X, N, M);
	    		kernelAoS_stream<<<gs, bs>>>(f, f_out, N, M);
	    	}
	    	//memory swap
	    	//kernel_copy<<<gs, bs>>>(f, f_out, i, N, M);
			  temp = f; //Me tiraba error por esto, pienso que es porque asignamos memoria de gpu en cpu
			  f = f_out;
			  f_out = temp;
        hipDeviceSynchronize();
	    }
      

	    hipEventRecord(ct2);
	    hipEventSynchronize(ct2);
	    hipEventElapsedTime(&dt, ct1, ct2);
	    std::cout << "Tiempo GPU: " << dt << "[ms]" << std::endl;
	    f_hostout = new int[M * N * X];
	    hipMemcpy(f_hostout, f, M * N * X * sizeof(int), hipMemcpyDeviceToHost);

	    if (i == 0){
    		Write_SoA(f_hostout, M, N, "initial.txt\0");
    	}
    	else{
    		Write_AoS(f_hostout, M, N, "initial.txt\0");
    	}

	    validar(f_hostout, N, M);

    	hipFree(f);
    	hipFree(temp);
    	hipFree(f_out);
    	delete[] f_host;
    	delete[] f_hostout;
	}
	return 0;
}